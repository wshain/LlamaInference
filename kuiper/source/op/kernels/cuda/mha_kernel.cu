#include "hip/hip_runtime.h"
#include <base/cuda_config.h>
#include <base/tick.h>
#include <tensor/tensor.h>
#include <cfloat>
#include <hipcub/hipcub.hpp>
#include "mha_kernel.cuh"
namespace kernel {
constexpr static int thread_num = 256;
/**
 * 在 GPU 上对一个数组执行 in-place Softmax 操作
 *
 * 功能: 对输入数组 x 执行 softmax: x[i] = exp(x[i] - max) / sum(exp(x[j] - max))
 * 使用 CUB 库进行高效的 block 内归约（reduce）
 *
 * 参数:
 *   x     - 输入/输出数组，shape: [size]，softmax 后结果写回原数组
 *   size  - 数组长度（通常是 seq_len，即注意力分数的序列长度）
 */
__device__ void softmax_gpu(float* __restrict__ x, int size) {
  int tid = threadIdx.x;
  int step = blockDim.x;

  // find max value (for numerical stability)
  // this should be FLT_MAX, not 0 !!!!
  // otherwise, the softmax may be occur nan when head_dim < 128 threads
  // ================= 第一步：找最大值（用于数值稳定）=================
  // 初始化 max_val：如果线程 ID 小于 size，则取 x[tid]，否则设为 -FLT_MAX
  // 这是为了避免越界线程影响最大值计算
  float max_val = tid < size ? x[tid] : -FLT_MAX;
  // 使用 grid-stride loop 在 block 内并行查找局部最大值
  // 每个线程从 tid + step 开始，每隔 step 个元素检查一次
  for (int i = tid + step; i < size; i += step) {
    if (x[i] > max_val) {
      max_val = x[i];
    }
  }
  // 使用 CUB 的 BlockReduce 进行 block 内归约（求最大值）
  using BlockReduce = hipcub::BlockReduce<float, thread_num>;  // 假设 thread_num 是编译期常量
  __shared__ BlockReduce::TempStorage temp;                 // 共享内存，用于 CUB 归约
  __shared__ float shared_val;                              // 用于存储归约结果（max 和 sum）
  // 执行归约操作：所有线程协作，找出整个 block 中的最大值
  max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
  // 只有主线程（threadIdx.x == 0）保存结果到共享内存
  if (threadIdx.x == 0) {
    shared_val = max_val;
  }
  __syncthreads();
  // 所有线程读取最大值
  max_val = shared_val;

  // ================= 第二步：计算 exp(x[i] - max_val) 并求和 =================
  float sum = 0.0f;
  for (int i = tid; i < size; i += step) {
    x[i] = expf(x[i] - max_val);  // in-place 计算 exp(x[i] - max)，提升数值稳定性
    sum += x[i];                  // 累加局部和
  }
  // 使用 BlockReduce 对 sum 进行 block 内归约（求和）
  sum = BlockReduce(temp).Sum(sum);
  // 主线程保存总和
  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  // 所有线程读取总和
  sum = shared_val;
  // ================= 第三步：归一化：x[i] /= sum =================
  for (int i = tid; i < size; i += step) {
    // 每个元素除以总和，完成 softmax
    x[i] /= sum;
  }
}

__global__ void multi_head_attention_kernel(int32_t pos, int32_t seq_len, float* query,
                                            float* score_ptr, float* output, float* key_cache,
                                            float* value_cache, int32_t kv_dim, int32_t kv_mul,
                                            int32_t head_num, int32_t head_size,
                                            int32_t layer_offset) {
  int head = blockIdx.x;
  if (head >= head_num) {
    return;
  }

  extern __shared__ float s_query_head[];
  float scale = 1.f / sqrtf(float(head_size));
  float* query_head = query + head * head_size;

  // 预加载query到共享内存
  for (int i = threadIdx.x; i < head_size; i += blockDim.x) {
    s_query_head[i] = query_head[i];
  }
  __syncthreads();

  float* score_head = score_ptr + head * seq_len;
  // head当前的注意力头索引，kv_mul用于gqa，head_size表示一个自注意力头的维度
  // kv_dim = head_size * head_num，多头自注意力情况下的key,value 维度
  // kv_dim = head_size * head_num / kv_num，GQA情况下的key,value 维度
  int head_offset = (head / kv_mul) * head_size;
  // 计算自注意力分数
  for (int t = threadIdx.x; t <= pos; t += blockDim.x) {
    float* key_head = key_cache + layer_offset + t * kv_dim + head_offset;

    float score = 0.0f;
    for (int i = 0; i < head_size; i += 4) {
      float4 key_val = *reinterpret_cast<float4*>(key_head + i);
      float4 query_val = *reinterpret_cast<float4*>(s_query_head + i);

      score += key_val.x * query_val.x + key_val.y * query_val.y + key_val.z * query_val.z +
               key_val.w * query_val.w;
    }

    score *= scale;
    score_head[t] = score;
  }
  __syncthreads();

  softmax_gpu(score_head, pos + 1);
  __syncthreads();

  float* output_head = output + head * head_size;
  // 使用自注意力分数对value矩阵加权
  for (int i = threadIdx.x; i < head_size; i += blockDim.x) {
    float value = 0.0f;
    for (int t = 0; t <= pos; t++) {
      float* value_head = value_cache + layer_offset + t * kv_dim + head_offset;
      float score = score_head[t];
      value += score * value_head[i];
    }
    output_head[i] = value;
  }
}

void mha_kernel_cu(int32_t pos, int32_t head_num, int32_t layer_index, int32_t seq_len,
                   int32_t kv_dim, int32_t kv_mul, int32_t head_size, const tensor::Tensor& mha_out,
                   const tensor::Tensor& query_tensor, const tensor::Tensor& score_tensor,
                   const tensor::Tensor& key_cache_tensor, const tensor::Tensor& value_cache_tensor,
                   base::DeviceType device_type, CudaConfig* config) {
  UNUSED(device_type);
  int32_t layer_offset = layer_index * seq_len * kv_dim;
  float* query = const_cast<float*>(query_tensor.ptr<float>());
  float* score = const_cast<float*>(score_tensor.ptr<float>());
  float* output = const_cast<float*>(mha_out.ptr<float>());

  float* key_cache = const_cast<float*>(key_cache_tensor.ptr<float>());
  float* value_cache = const_cast<float*>(value_cache_tensor.ptr<float>());

  hipStream_t stream = config->stream;
  multi_head_attention_kernel<<<head_num, thread_num, head_size * sizeof(float), stream>>>(
      pos, seq_len, query, score, output, key_cache, value_cache, kv_dim, kv_mul, head_num,
      head_size, layer_offset);
}

}  // namespace kernel